#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>


__global__ void kernel(int* da,int n1){
    int k=threadIdx.x,temp;
    for(int j=0;j<n1;j++){
        temp=da[k*n1+j];
        for(int i=0;i<k;i++)
            da[k*n1+j]*=temp;
    }
}

int main(){
    int m1,n1;
    printf("Enter m1,n1 : ");
    scanf("%d %d",&m1,&n1);

    int a[m1][n1];
    printf("Enter mat : ");
    for(int i=0;i<m1;i++)
        for(int j=0;j<n1;j++)
            scanf("%d",&a[i][j]);

    int* da;
    hipMalloc((void **)&da, (m1*n1) * sizeof(int));
    hipMemcpy(da,a, m1*n1 * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(1, 1, 1);
    dim3 blk(m1, 1, 1);

    kernel<<<grid, blk>>>(da,n1);

    hipMemcpy(a, da, m1*n1*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0;i<m1;i++){
        for(int j=0;j<n1;j++)
            printf("%d\t",a[i][j]);
        printf("\n");
    }
    hipFree(da);
}
