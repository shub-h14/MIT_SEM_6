#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>


__global__ void kernel(int* da,int n1){
    int k=threadIdx.x;
    if(k==0||k==blockDim.x-1)
        return;
    printf("%d\n",(int)powf(10,0));
    for(int j=1;j<n1-1;j++){
        int num=da[k*n1+j];
        int out=0,pow=0,place=1;
        while(num){
            int tt=1;
            if(num%2)
                tt=0;
            out=tt*place+out;
            num/=2;
            place*=10;
            printf("%d - %d - %d\n",k,j,out);
        }
        da[k*n1+j]=out;
    }
}

int main(){
    int m1,n1;
    printf("Enter m1,n1 : ");
    scanf("%d %d",&m1,&n1);

    int a[m1][n1];
    printf("Enter mat : ");
    for(int i=0;i<m1;i++)
        for(int j=0;j<n1;j++)
            scanf("%d",&a[i][j]);

    int* da;
    hipMalloc((void **)&da, (m1*n1) * sizeof(int));
    hipMemcpy(da,a, m1*n1 * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(1, 1, 1);
    dim3 blk(m1, 1, 1);

    kernel<<<grid, blk>>>(da,n1);

    hipMemcpy(a, da, m1*n1*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0;i<m1;i++){
        for(int j=0;j<n1;j++)
            printf("%d\t",a[i][j]);
        printf("\n");
    }
    hipFree(da);
}
