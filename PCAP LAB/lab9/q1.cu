//Write a program in CUDA to perform parallel Sparse Matrix - Vector multiplication 
//using compresssed sparse row (CSR) storage format. Represent the input sparse 
//matrix in CSR format in the host code. 

#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>


__global__ void kernel(int* da,int* ca,int* ra,int* db,int* o){
    int k=threadIdx.x;
    int s=ra[k];
    int e=ra[k+1];
    for(int j=s;j<e;j++){
        o[k]+=da[j]*db[ca[j]];
    }
}

int main(){
    int m1,n1;
    printf("Enter m1,n1 : ");
    scanf("%d %d",&m1,&n1);

    int a[m1][n1],b[n1];
    int c=0;
    printf("Enter mat : ");
    for(int i=0;i<m1;i++){
        for(int j=0;j<n1;j++){
            scanf("%d",&a[i][j]);
            if(a[i][j]!=0)
                c++;
        }
    }
    
    printf("Enter vec : ");
    for(int i=0;i<n1;i++)
        scanf("%d",&b[i]);
    
    int data[c],col[c],row[m1+1];
    int k1=0,k2=0;
    row[m1]=c;
    for(int i=0;i<m1;i++){
        row[k2++]=k1;
        for(int j=0;j<n1;j++){
            if(a[i][j]!=0){
                data[k1]=a[i][j];
                col[k1++]=j;
            }
        }
    }

    int* da,*ca,*ra,*db,*o;
    hipMalloc((void **)&da, c * sizeof(int));
    hipMalloc((void **)&ca, c * sizeof(int));
    hipMalloc((void **)&ra, (m1+1) * sizeof(int));
    hipMalloc((void **)&db, n1 * sizeof(int));
    hipMalloc((void **)&o, m1*sizeof(int));

    hipMemcpy(da,data, c * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ca,col, c * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ra,row, (m1+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db,b, n1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(o, 0, m1 * sizeof(int));


    dim3 grid(1, 1, 1);
    dim3 blk(m1, 1, 1);

    kernel<<<grid, blk>>>(da,ca, ra, db,o);

    int oh[m1];
    hipMemcpy(oh, o, m1*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0;i<m1;i++)
        printf("%d\t",oh[i]);
    printf("\n");
    hipFree(da);
    hipFree(ca);
    hipFree(ra);
    hipFree(db);
    hipFree(o);
}
