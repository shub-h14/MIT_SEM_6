#include <stdio.h>
#include <hip/hip_runtime.h>
 
#define N 12          
#define BLOCK_SIZE 4  
 
__global__ void Scan(int *d_in, int *d_out, int *d_blockSums, int n) {
    __shared__ int temp[BLOCK_SIZE];
 
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + tid;
   
    temp[tid] = (gid < n) ? d_in[gid] : 0;
    __syncthreads();
 
    
    for (int offset = 1; offset <=tid; offset *= 2) {
        int val = temp[tid - offset] ;
        __syncthreads();
        temp[tid] += val;
        __syncthreads();
    }
 
    if (tid == BLOCK_SIZE - 1) {
        d_blockSums[blockIdx.x] = temp[tid];
        printf("%d -%d\n ",  d_blockSums[blockIdx.x], blockIdx.x  );
    }
    
 
    if (gid < n) d_out[gid] = temp[tid];
}
__global__ void addBlockOffsets(int *d_out, int *d_blockSums, int n) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
     int i;
    if (blockIdx.x > 0 && gid < n) {
      for(i=0;i<blockIdx.x;i++)
        d_out[gid] += d_blockSums[i];
    }
}
 
int main() {
    int h_in[N] = {1, 2, 3, 4, 1, 1, 1, 1, 2, 2, 2, 2};
    int h_out[N];
 
    int *d_in, *d_out, *d_blockSums;
    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
 
    hipMalloc(&d_in, N * sizeof(int));
    int err=hipMalloc(&d_out, N * sizeof(int));
    if(err!=hipSuccess)
    printf("error\n");
    hipMalloc(&d_blockSums, numBlocks * sizeof(int));
 
    hipMemcpy(d_in, h_in, N * sizeof(int), hipMemcpyHostToDevice);
 
    Scan<<<numBlocks, BLOCK_SIZE>>>(d_in, d_out, d_blockSums, N);
 
    addBlockOffsets<<<numBlocks, BLOCK_SIZE>>>(d_out, d_blockSums, N);
 
    hipMemcpy(h_out, d_out, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("Prefix Sum: ");
    for (int i = 0; i < N; i++) printf("%d ", h_out[i]);
    printf("\n");
 
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_blockSums);
    return 0;
}
