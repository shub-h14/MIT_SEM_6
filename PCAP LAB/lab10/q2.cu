#include "hip/hip_runtime.h"

#include<stdio.h>
__device__ int findgtid()
{
    int tnb,bng,ntpd;
    ntpd=blockDim.x*blockDim.y*blockDim.z;
    tnb=threadIdx.x+blockDim.x*threadIdx.y+blockDim.x*blockDim.y*threadIdx.z;
    bng=blockIdx.x+gridDim.x*blockIdx.y+gridDim.x*gridDim.y*blockIdx.z;
    return bng*ntpd+tnb;
}
__global__ void mykernel(int *A,int *B,int* C,int m)
{
    int gtid=findgtid();
    __shared__ int sA[10],sC[5];
    sA[gtid]=A[gtid];
    if(gtid<m) sC[gtid]=C[gtid];
    __syncthreads();
    int sum=0;
    int sp=gtid-m/2;
    for(int i=0;i<m;i++)
    {
        if(sp+i>=0 && sp+i<blockDim.x)sum+=sA[sp+i]*sC[i];
    }
    B[gtid]=sum;
}

int main()
{
    int *A,*B,*C;
    int n=10,m=5;
    int *d_A,*d_B,*d_C;
    size_t s=sizeof(int) * n;
    size_t m1=sizeof(int)*m;
    A=(int *)malloc(s);
    B=(int *)malloc(s);
    C=(int *)malloc(m1);
    hipMalloc(&d_A,s);
    hipMalloc(&d_B,s);
    hipMalloc(&d_C,m1);
    printf("Enter Array:");
    for(int i=0;i<n;i++)scanf("%d",&A[i]);
    printf("Enter Mask:");
    for(int i=0;i<m;i++)scanf("%d",&C[i]);
    hipMemcpy(d_A,A,s,hipMemcpyHostToDevice);
    hipMemcpy(d_C,C,m1,hipMemcpyHostToDevice);
    mykernel<<<1,n>>>(d_A,d_B,d_C,m);
    hipMemcpy(B,d_B,s,hipMemcpyDeviceToHost);
    printf("Final vector:");
    for(int i=0;i<n;i++)printf("%d ",B[i]);
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
