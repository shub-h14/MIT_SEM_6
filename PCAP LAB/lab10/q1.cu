#include "hip/hip_runtime.h"

#include<stdio.h>
__global__ void matmul(int *A,int *B,int *C,int n,int q){
    int row=threadIdx.y+blockDim.y*blockIdx.y;
    int col=threadIdx.x+blockDim.x*blockIdx.x;
    int sum=0;
    for(int i=0;i<n;i++)
    {
        sum+=A[row*n+i]*B[i*q+col];
    }
    C[row*q+col]=sum;
}
int main(){
    int m,n,p,q;
    int *d_A,*d_B,*d_C;
    printf("Enter m and n: ");
    scanf("%d %d",&m,&n);
    printf("Enter p and q: ");
    scanf("%d %d",&p,&q);
    int A[m][n],B[p][q],C[m][q];
    printf("Enter A:");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            scanf("%d",&A[i][j]);
        }
    }
    printf("Enter B:");
    for(int i=0;i<p;i++){
        for(int j=0;j<q;j++){
            scanf("%d",&B[i][j]);
        }
    }
    hipMalloc(&d_A,m*n*sizeof(int));
    hipMalloc(&d_B,p*q*sizeof(int));
    hipMalloc(&d_C,m*q*sizeof(int));
    hipMemcpy(d_A,A[0],m*n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B[0],p*q*sizeof(int),hipMemcpyHostToDevice);
    dim3 blck(2,2,1);
    dim3 grid(ceil(q/2.0),ceil(m/2.0));
    matmul<<<grid,blck>>>(d_A,d_B,d_C,n,q);
    hipMemcpy(C[0],d_C,m*q*sizeof(int),hipMemcpyDeviceToHost);
    printf("\nFinal Matrix:\n");
    for(int i=0;i<m;i++){
        for(int j=0;j<q;j++){
        printf("%3d ",C[i][j]);
        }
        printf("\n");
    }
}   
