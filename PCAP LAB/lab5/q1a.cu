//write a program in cuda to add two vectors of length N using
// a) block size as N 

#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add_vec(int* da, int* db, int* dc) {
    int index = threadIdx.x;  
    dc[index] = da[index] + db[index];
}

int main() {
    int n;
    printf("Length of the vector: ");
    scanf("%d", &n);

    if (n > 1024) {
        printf("Error: Max threads per block is 1024. Try n <= 1024.\n");
        return 1;
    }

    int *a = new int[n];
    int *b = new int[n];
    int *c = new int[n];
    int *da, *db, *dc;

    hipMalloc((void**)&da, n * sizeof(int));
    hipMalloc((void**)&db, n * sizeof(int));
    hipMalloc((void**)&dc, n * sizeof(int));

    printf("Enter vector one: ");
    for (int i = 0; i < n; i++)
        scanf("%d", &a[i]);

    printf("Enter vector two: ");
    for (int i = 0; i < n; i++)
        scanf("%d", &b[i]);

    hipMemcpy(da, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, b, n * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(1, 1, 1);
    dim3 blk(n, 1, 1);
    add_vec<<<grid, blk>>>(da, db, dc);

    hipMemcpy(c, dc, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result vector: ");
    for (int i = 0; i < n; i++)
        printf("%d\t", c[i]);
    printf("\n");

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}

