//Implement a cuda program to add two vectors of length N 
//by keeping the number of threads per block as 256 (constant) 
//and vary the number of blocks to handle N elements.

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define THREADS_PER_BLOCK 256

__global__ void add_vec(int* a, int* b, int* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        c[idx] = a[idx] + b[idx];
}

int main() {
    int n;
    printf("Enter length of vectors: ");
    scanf("%d", &n);

    int* h_a = (int*)malloc(n * sizeof(int));
    int* h_b = (int*)malloc(n * sizeof(int));
    int* h_c = (int*)malloc(n * sizeof(int));

    printf("Enter vector one:\n");
    for (int i = 0; i < n; i++)
        scanf("%d", &h_a[i]);

    printf("Enter vector two:\n");
    for (int i = 0; i < n; i++)
        scanf("%d", &h_b[i]);

    int* d_a, * d_b, * d_c;
    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_b, n * sizeof(int));
    hipMalloc((void**)&d_c, n * sizeof(int));

    hipMemcpy(d_a, h_a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(int), hipMemcpyHostToDevice);

    int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    add_vec<<<blocks, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();

    hipMemcpy(h_c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Resultant vector:\n");
    for (int i = 0; i < n; i++)
        printf("%d ", h_c[i]);
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}

