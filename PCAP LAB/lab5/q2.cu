//Implement a cuda program to add two vectors of length N 
//by keeping the number of threads per block as 256 (constant) 
//and vary the number of blocks to handle N elements.

#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void add_vec(int*da,int*db,int*dc){
    int index=blockIdx.x*blockDim.x+threadIdx.x;
    dc[index]=da[index]+db[index];
}

int main(){
    int n;
    printf("Length of the vector : ");
    scanf("%d",&n);

    int gridn=ceil(n/256);
    int a[n],b[n],c[n];
    int *da,*db,*dc;

    hipMalloc((void **)&da,n*sizeof(int));
    hipMalloc((void **)&db,n*sizeof(int));
    hipMalloc((void **)&dc,n*sizeof(int));

    printf("Enter vector one : ");
    for(int i=0;i<n;i++)
        a[i]=i;

    printf("Enter vector two : ");
    for(int i=0;i<n;i++)
        b[i]=i;
    
    hipMemcpy(da,a,n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(db,b,n*sizeof(int),hipMemcpyHostToDevice);

    dim3 grid(gridn,1,1);
    dim3 blk(256,1,1);

    add_vec<<<grid,blk>>>(da,db,dc);
    hipMemcpy(c,dc,n*sizeof(int),hipMemcpyDeviceToHost);

    for(int i=0;i<n;i++)
        printf("%d\n",c[i]);
    printf("\n");
    hipFree(da);
    hipFree(db);
    hipFree(dc);
}

