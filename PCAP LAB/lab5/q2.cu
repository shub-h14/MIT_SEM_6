//Implement a CUDA program to add two vectors of length N 
//by keeping the number of threads per block as 256 (constant) 
//and vary the number of blocks to handle N elements.

#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add_vec(int *da, int *db, int *dc) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < gridDim.x * blockDim.x) {
        dc[index] = da[index] + db[index];
    }
}

int main() {
    int n;
    printf("Length of the vector: ");
    scanf("%d", &n);

    int *a = new int[n];
    int *b = new int[n];
    int *c = new int[n];
    int *da, *db, *dc;

    hipMalloc((void**)&da, n * sizeof(int));
    hipMalloc((void**)&db, n * sizeof(int));
    hipMalloc((void**)&dc, n * sizeof(int));

    printf("Enter vector one: ");
    for (int i = 0; i < n; i++)
        scanf("%d", &a[i]);

    printf("Enter vector two: ");
    for (int i = 0; i < n; i++)
        scanf("%d", &b[i]);

    hipMemcpy(da, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, b, n * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    dim3 grid(blocks, 1, 1);
    dim3 blk(threadsPerBlock, 1, 1);

    add_vec<<<grid, blk>>>(da, db, dc);

    hipMemcpy(c, dc, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result vector:\n");
    for (int i = 0; i < n; i++)
        printf("%d\n", c[i]);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}
