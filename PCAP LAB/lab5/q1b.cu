//write a program in cuda to add two vectors of length N using 
//b)N threads 

#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add_vec(int* da, int* db, int* dc) {
    int index = threadIdx.x;
    dc[index] = da[index] + db[index];
}

int main() {
    int n;
    printf("Length of the vector (<= 1024): ");
    scanf("%d", &n);

    if (n > 1024) {
        printf("Error: Maximum threads per block is 1024. Please enter n <= 1024.\n");
        return -1;
    }

    int* a = new int[n];
    int* b = new int[n];
    int* c = new int[n];

    printf("Enter vector one: ");
    for (int i = 0; i < n; i++) {
        scanf("%d", &a[i]);
    }

    printf("Enter vector two: ");
    for (int i = 0; i < n; i++) {
        scanf("%d", &b[i]);
    }

    int* da, * db, * dc;

    hipMalloc((void**)&da, n * sizeof(int));
    hipMalloc((void**)&db, n * sizeof(int));
    hipMalloc((void**)&dc, n * sizeof(int));

    hipMemcpy(da, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, b, n * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(1);
    dim3 blk(n);
    add_vec<<<grid, blk>>>(da, db, dc);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
        return -1;
    }

    hipDeviceSynchronize();

    hipMemcpy(c, dc, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Resultant vector: ");
    for (int i = 0; i < n; i++) {
        printf("%d\t", c[i]);
    }
    printf("\n");

    delete[] a;
    delete[] b;
    delete[] c;
    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return 0;
}

