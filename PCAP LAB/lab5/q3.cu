//write a program in cuda to process a 1d array containg angles
//in radians to generate sine of the angles in 
//the output array.Use approariate functions.

#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void add_vec(float*da,float*dc){
    int index=blockIdx.x*blockDim.x+threadIdx.x;
    dc[index]=sinf(da[index]);
}

int main(){
    int n;
    printf("Length of the vector : ");
    scanf("%d",&n);

    float a[n],c[n];
    float *da,*dc;

    hipMalloc((void **)&da,n*sizeof(float));
    hipMalloc((void **)&dc,n*sizeof(float));

    printf("Enter vector one : ");
    for(int i=0;i<n;i++)
        scanf("%f",&a[i]);
    
    hipMemcpy(da,a,n*sizeof(int),hipMemcpyHostToDevice);

    dim3 grid(n,1,1);
    dim3 blk(1,1,1);

    add_vec<<<grid,blk>>>(da,dc);
    hipMemcpy(c,dc,n*sizeof(int),hipMemcpyDeviceToHost);

    for(int i=0;i<n;i++)
        printf("%f\t",c[i]);
    printf("\n");
    hipFree(da);
    hipFree(dc);
}
