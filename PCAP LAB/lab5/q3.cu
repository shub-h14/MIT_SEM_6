//write a program in cuda to process a 1d array containg angles
//in radians to generate sine of the angles in 
//the output array.Use approariate functions.

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define THREADS_PER_BLOCK 256
__global__ void compute_sine(float* angles, float* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        output[idx] = sinf(angles[idx]);  // Use sinf for float
}
int main() {
    int n;
    printf("Enter number of angles: ");
    scanf("%d", &n);

    float* h_angles = (float*)malloc(n * sizeof(float));
    float* h_output = (float*)malloc(n * sizeof(float));

    printf("Enter %d angles in radians:\n", n);
    for (int i = 0; i < n; i++)
        scanf("%f", &h_angles[i]);

    float *d_angles, *d_output;
    hipMalloc((void**)&d_angles, n * sizeof(float));
    hipMalloc((void**)&d_output, n * sizeof(float));

    hipMemcpy(d_angles, h_angles, n * sizeof(float), hipMemcpyHostToDevice);

    int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    compute_sine<<<blocks, THREADS_PER_BLOCK>>>(d_angles, d_output, n);
    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    printf("Sine values:\n");
    for (int i = 0; i < n; i++)
        printf("sin(%.4f) = %.4f\n", h_angles[i], h_output[i]);

    hipFree(d_angles);
    hipFree(d_output);
    free(h_angles);
    free(h_output);

    return 0;
}

