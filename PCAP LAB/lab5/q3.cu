// CUDA program to process a 1D array containing angles in radians
// and generate sine of the angles in the output array using sinf().

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h> // for sinf()

__global__ void compute_sine(float* da, float* dc, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        dc[index] = sinf(da[index]);
    }
}

int main() {
    int n;
    printf("Length of the vector: ");
    scanf("%d", &n);

    float* a = new float[n];
    float* c = new float[n];
    float* da, * dc;

    hipMalloc((void**)&da, n * sizeof(float));
    hipMalloc((void**)&dc, n * sizeof(float));

    printf("Enter the angles in radians:\n");
    for (int i = 0; i < n; i++)
        scanf("%f", &a[i]);

    hipMemcpy(da, a, n * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    dim3 grid(blocksPerGrid, 1, 1);
    dim3 blk(threadsPerBlock, 1, 1);

    compute_sine<<<grid, blk>>>(da, dc, n);
    hipMemcpy(c, dc, n * sizeof(float), hipMemcpyDeviceToHost);

    printf("Sine values:\n");
    for (int i = 0; i < n; i++)
        printf("sin(%f) = %f\n", a[i], c[i]);

    hipFree(da);
    hipFree(dc);
    delete[] a;
    delete[] c;

    return 0;
}
