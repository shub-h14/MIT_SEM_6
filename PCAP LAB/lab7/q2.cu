// Write a CUDA program that reads a string S and produces the string RS as follows:
// Input string S: PCAP               
// Output string RS: PCAPPCAPCP

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void kernel(char* sin, int* sin_len, char* sout) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    int si = 0;
    for (int i = 0; i < idx; i++) {
        si += (*sin_len) - i;
    }

    int chars_to_copy = (*sin_len) - idx;
    for (int i = 0; i < chars_to_copy; i++) {
        sout[si + i] = sin[i];
    }
}

int main() {
    char sin[100], sout[100];
    printf("Enter string: ");
    scanf("%s", sin);

    int sin_len = strlen(sin);
    int sout_len = 0;

    for (int i = 0; i < sin_len; i++) {
        sout_len += sin_len - i;
    }

    char* d_sin;
    int* d_sin_len;
    char* d_sout;

    hipMalloc((void**)&d_sin, sin_len * sizeof(char));
    hipMalloc((void**)&d_sin_len, sizeof(int));
    hipMalloc((void**)&d_sout, (sout_len + 1) * sizeof(char));  

    hipMemcpy(d_sin, sin, sin_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_sin_len, &sin_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_sout, 0, (sout_len + 1) * sizeof(char)); 

    kernel<<<1, sin_len>>>(d_sin, d_sin_len, d_sout);

    hipMemcpy(sout, d_sout, (sout_len + 1) * sizeof(char), hipMemcpyDeviceToHost);
    sout[sout_len] = '\0';  

    printf("Output string RS: %s\n", sout);

    hipFree(d_sin);
    hipFree(d_sin_len);
    hipFree(d_sout);

    return 0;
}
