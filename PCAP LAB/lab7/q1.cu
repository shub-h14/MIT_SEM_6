//Write a program in CUDA to count the number of times a given word is repeated in a sentence.

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void word_count_kernel(char* str, char* key, int str_len, int key_len, int* word_indices, int word_count, int* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= word_count) return;

    int si = word_indices[idx];
    int ei = (idx + 1 < word_count) ? word_indices[idx + 1] : str_len;
    
    bool is_equal = true;
    int i1 = si, i2 = 0;

    while (i1 < ei && i2 < key_len) {
        if (str[i1] != key[i2]) {
            is_equal = false;
            break;
        }
        i1++;
        i2++;
    }

    if (is_equal && i2 == key_len) {
        atomicAdd(result, 1);
    }
}

int main() {
    char str[100], key[20];
    printf("Enter string: ");
    scanf("%[^\n]s", str);
    printf("Enter key: ");
    scanf("%s", key);

    int str_len = strlen(str), key_len = strlen(key);

    int word_count = 0;
    int* word_indices = (int*)malloc((str_len / 2 + 1) * sizeof(int));  

    word_indices[word_count++] = 0;  

    for (int i = 0; i < str_len; i++) {
        if (str[i] == ' ') {
            word_indices[word_count++] = i + 1;
        }
    }

    char* d_str;
    char* d_key;
    int* d_word_indices;
    int* d_result;
    
    hipMalloc((void**)&d_str, str_len * sizeof(char));
    hipMalloc((void**)&d_key, key_len * sizeof(char));
    hipMalloc((void**)&d_word_indices, word_count * sizeof(int));
    hipMalloc((void**)&d_result, sizeof(int));
    
    hipMemcpy(d_str, str, str_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_key, key, key_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word_indices, word_indices, word_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_result, 0, sizeof(int));

    int grid_size = (word_count + 256 - 1) / 256;

    word_count_kernel<<<grid_size, 256>>>(d_str, d_key, str_len, key_len, d_word_indices, word_count, d_result);
    int result = 0;
    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("Input String: %s\n", str);
    printf("Key: %s\n", key);
    printf("Total occurrences of '%s': %d\n", key, result);

    hipFree(d_str);
    hipFree(d_key);
    hipFree(d_word_indices);
    hipFree(d_result);
    free(word_indices);
    return 0;
}
