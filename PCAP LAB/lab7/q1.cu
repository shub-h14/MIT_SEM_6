//Write a program in CUDA to count the number of times a given word is repeated in a sentence.

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void word_count_kernel(char* str, char* key, int str_len, int key_len, int* word_indices, int word_count, int* result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= word_count)
        return;

    int si = word_indices[idx];                             // Start index of the word
    int ei = (idx + 1 < word_count) ? word_indices[idx + 1] - 1 : str_len;  // End index

    bool is_equal = true;
    int i1 = si, i2 = 0;

    while (i1 < ei && i2 < key_len) {
        if (str[i1] != key[i2]) {
            is_equal = false;
            break;
        }
        i1++;
        i2++;
    }

    // If the key length is fully matched and reached word end
    if (is_equal && i2 == key_len && i1 == ei) {
        atomicAdd(result, 1);  // Safe atomic update from multiple threads
    }
}

int main() {
    char str[100], key[20];

    printf("Enter string: ");
    scanf(" %[^\n]s", str);

    printf("Enter key: ");
    scanf("%s", key);

    int str_len = strlen(str);
    int key_len = strlen(key);

    int word_count = 0;
    int* word_indices = (int*)malloc((str_len / 2 + 1) * sizeof(int));

    word_indices[word_count++] = 0;

    // Store indices of where each word starts
    for (int i = 0; i < str_len; i++) {
        if (str[i] == ' ') {
            word_indices[word_count++] = i + 1;
        }
    }

    // CUDA memory declarations
    char *d_str, *d_key;
    int *d_word_indices, *d_result;

    hipMalloc((void**)&d_str, str_len * sizeof(char));
    hipMalloc((void**)&d_key, key_len * sizeof(char));
    hipMalloc((void**)&d_word_indices, word_count * sizeof(int));
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_str, str, str_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_key, key, key_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word_indices, word_indices, word_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_result, 0, sizeof(int));

    dim3 blk(256, 1, 1);
    dim3 grid((word_count + 255) / 256, 1, 1);

    word_count_kernel<<<grid, blk>>>(d_str, d_key, str_len, key_len, d_word_indices, word_count, d_result);

    int result = 0;
    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("Input String: %s\n", str);
    printf("Key: %s\n", key);
    printf("Total occurrences of '%s': %d\n", key, result);

    // Free memory
    hipFree(d_str);
    hipFree(d_key);
    hipFree(d_word_indices);
    hipFree(d_result);
    free(word_indices);

    return 0;
}
