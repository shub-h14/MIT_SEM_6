//Write a program in cuda which performs convolution operation on one-dimensional input 
//array N of size width using a mask array M of size_width to produce the resultant 
//one-dimensional array P of size width.

#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void kernel(float*da,float*db,float*dc,int mw,int w){
    int i=blockIdx.x*blockDim.x+threadIdx.x;

    int s=i-(mw)/2;
    float pv=0;
    for(int j=0;j<w;j++){
        if(s+j>=0&&s+j<w){
            pv+=da[s+j]*db[j];
        }
    }
    dc[i]=pv;
}

int main(){
    int n1,n2;

    printf("Length of the vector : ");
    scanf("%d",&n1);

    printf("Enter the length of mask : ");
    scanf("%d",&n2);


    float a[n1],b[n2],c[n1];
    float *da,*db,*dc;

    hipMalloc((void **)&da,n1*sizeof(float));
    hipMalloc((void **)&db,n2*sizeof(float));
    hipMalloc((void **)&dc,n1*sizeof(float));

    printf("Enter vector one : ");
    for(int i=0;i<n1;i++)
        scanf("%f",&a[i]);

    printf("Enter vector two (aka mask) : ");
    for(int i=0;i<n2;i++)
        scanf("%f",&b[i]);
    
    hipMemcpy(da,a,n1*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(db,b,n2*sizeof(float),hipMemcpyHostToDevice);

    dim3 grid(n1,1,1);
    dim3 blk(1,1,1);

    kernel<<<grid,blk>>>(da,db,dc,n2,n1);
    hipMemcpy(c,dc,n1*sizeof(float),hipMemcpyDeviceToHost);

    for(int i=0;i<n1;i++)
        printf("%f\t",c[i]);
    printf("\n");
    hipFree(da);
    hipFree(db);
    hipFree(dc);
}
