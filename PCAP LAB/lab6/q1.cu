// Write a program in CUDA which performs convolution operation on a 1D input 
// array N of size 'width' using a mask array M of size 'maskWidth' 
// to produce the resultant 1D array P of the same size.

#include "hip/hip_runtime.h"

#include <stdio.h>

// CUDA kernel to perform 1D convolution
__global__ void convolve_1d(float* da, float* db, float* dc, int maskWidth, int width) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int start = i - (maskWidth / 2);
    float sum = 0.0f;

    for (int j = 0; j < maskWidth; j++) {
        int idx = start + j;
        if (idx >= 0 && idx < width) {
            sum += da[idx] * db[j];
        }
    }

    dc[i] = sum;
}

int main() {
    int n1, n2;

    printf("Length of the input array: ");
    scanf("%d", &n1);

    printf("Length of the mask array: ");
    scanf("%d", &n2);

    float a[n1], b[n2], c[n1];
    float *da, *db, *dc;

    hipMalloc((void**)&da, n1 * sizeof(float));
    hipMalloc((void**)&db, n2 * sizeof(float));
    hipMalloc((void**)&dc, n1 * sizeof(float));

    printf("Enter input array (N): ");
    for (int i = 0; i < n1; i++)
        scanf("%f", &a[i]);

    printf("Enter mask array (M): ");
    for (int i = 0; i < n2; i++)
        scanf("%f", &b[i]);

    hipMemcpy(da, a, n1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, b, n2 * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(n1, 1, 1);  // One thread per output element
    dim3 blk(1, 1, 1);

    convolve_1d<<<grid, blk>>>(da, db, dc, n2, n1);

    hipMemcpy(c, dc, n1 * sizeof(float), hipMemcpyDeviceToHost);

    printf("Resultant array (P):\n");
    for (int i = 0; i < n1; i++)
        printf("%f\t", c[i]);
    printf("\n");

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return 0;
}
