//Write a program in CUDA to perform selection sort in parallel.

#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void add_vec(int*da,int*dc,int s){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int k=0;
    for(int j=0;j<s;j++){
        if((da[j]<da[i])||(da[j]==da[i]&&j>i))
            k++;
    }
    dc[k]=da[i];
}

int main(){
    int n;
    printf("Length of the vector : ");
    scanf("%d",&n);

    int a[n],c[n];
    int *da,*dc;

    hipMalloc((void **)&da,n*sizeof(int));
    hipMalloc((void **)&dc,n*sizeof(int));

    printf("Enter vector one : ");
    for(int i=0;i<n;i++)
        scanf("%d",&a[i]);
    
    hipMemcpy(da,a,n*sizeof(int),hipMemcpyHostToDevice);

    dim3 grid(n,1,1);
    dim3 blk(1,1,1);

    add_vec<<<grid,blk>>>(da,dc,n);
    hipMemcpy(c,dc,n*sizeof(int),hipMemcpyDeviceToHost);

    for(int i=0;i<n;i++)
        printf("%d\t",c[i]);
    printf("\n");
    hipFree(da);
    hipFree(dc);
}
