//Write a program in CUDA to perform selection sort in parallel.

#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void selection_sort_parallel(int* da, int* dc, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int position = 0;

    // Count how many elements are smaller than da[i]
    // or equal with a smaller index to maintain order for duplicates
    for (int j = 0; j < size; j++) {
        if ((da[j] < da[i]) || (da[j] == da[i] && j < i)) {
            position++;
        }
    }

    // Place element at its correct sorted position
    dc[position] = da[i];
}

int main() {
    int n;
    printf("Length of the vector : ");
    scanf("%d", &n);

    int a[n], c[n];
    int *da, *dc;

    hipMalloc((void**)&da, n * sizeof(int));
    hipMalloc((void**)&dc, n * sizeof(int));

    printf("Enter vector elements : ");
    for (int i = 0; i < n; i++)
        scanf("%d", &a[i]);

    hipMemcpy(da, a, n * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(n, 1, 1);
    dim3 blk(1, 1, 1);

    selection_sort_parallel<<<grid, blk>>>(da, dc, n);
    hipMemcpy(c, dc, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted vector : ");
    for (int i = 0; i < n; i++)
        printf("%d\t", c[i]);
    printf("\n");

    hipFree(da);
    hipFree(dc);

    return 0;
}
