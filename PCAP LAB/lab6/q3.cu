//Write a program in CUDA to perform odd even transposition sort in parallel.

#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void add_vec(int*da,int s,int k){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int temp=i*2;
    if(k==0){
        temp++;
    }
    if(temp+1>s-1)
        return;
    if(da[temp]>da[temp+1]){
        da[temp]=da[temp]^da[temp+1];
        da[temp+1]=da[temp]^da[temp+1];
        da[temp]=da[temp+1]^da[temp];
    }
}

int main(){
    int n;
    printf("Length of the vector : ");
    scanf("%d",&n);

    int a[n];
    int *da;

    hipMalloc((void **)&da,n*sizeof(int));

    printf("Enter vector one : ");
    for(int i=0;i<n;i++)
        scanf("%d",&a[i]);
    
    hipMemcpy(da,a,n*sizeof(int),hipMemcpyHostToDevice);

    dim3 grid(n/2,1,1);
    dim3 blk(1,1,1);
    for(int i=1;i<n+1;i++){
        add_vec<<<grid,blk>>>(da,n,i%2);
    }
    hipMemcpy(a,da,n*sizeof(int),hipMemcpyDeviceToHost);

    for(int i=0;i<n;i++)
        printf("%d\t",a[i]);
    printf("\n");
    hipFree(da);
}
