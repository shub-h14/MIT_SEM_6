//Write a program in CUDA to multiply two matrices for the following specifications:
//a. Each row of resultant matrix to be computed by one thread.
//b. Each column of resultant matrix to be computed by one thread.
//c. Each element of resultant matrix to be computed by one thread.

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void multiply_matrices_row(int *a, int *b, int *c, int n, int m, int k)
{
        int row = blockIdx.x;
        for (int col = 0; col < k; col++)
        {
                int sum = 0;
                for (int i = 0; i < m; i++)
                {
                        sum += a[row * m + i] * b[i * k + col];
                }
                c[row * k + col] = sum;
        }
}

__global__ void multiply_matrices_col(int *a, int *b, int *c, int n, int m, int k)
{
        int col = blockIdx.x;
        for (int row = 0; row < n; row++)
        {
                int sum = 0;
                for (int i = 0; i < m; i++)
                {
                        sum += a[row * m + i] * b[i * k + col];
                }
                c[row * k + col] = sum;
        }
}

__global__ void multiply_matrices_elem(int *a, int *b, int *c, int n, int m, int k)
{
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if (tid < n * k)
        {
                int row = tid / k;
                int col = tid % k;
                int sum = 0;
                for (int i = 0; i < m; i++)
                {
                        sum += a[row * m + i] * b[i * k + col];
                }
                c[row * k + col] = sum;
        }
}

int main()
{
        int *a, *b, *c;
        int *d_a, *d_b, *d_c;
        int n, m, k;

        printf("Enter dimensions of matrix A (n x m): ");
        scanf("%d%d", &n, &m);
        printf("Enter dimensions of matrix B (m x k): ");
        scanf("%d%d", &m, &k);

        int size_a = n * m * sizeof(int);
        int size_b = m * k * sizeof(int);
        int size_c = n * k * sizeof(int);
        a = (int *)malloc(size_a);
        b = (int *)malloc(size_b);
        c = (int *)malloc(size_c);
        hipMalloc(&d_a, size_a);
        hipMalloc(&d_b, size_b);
        hipMalloc(&d_c, size_c);

        printf("Enter matrix A: \n");
        for (int i = 0; i < n; i++)
        {
                for (int j = 0; j < m; j++)
                {
                        scanf("%d", &a[i * m + j]);
                }
        }
        printf("Enter matrix B: \n");
        for (int i = 0; i < m; i++)
        {
                for (int j = 0; j < k; j++)
                {
                        scanf("%d", &b[i * k + j]);
                }
        }
        hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);

        dim3 grid_size_row(n, 1, 1);
        dim3 block_size_row(1, 1, 1);
        dim3 grid_size_col(k, 1, 1);
        dim3 block_size_col(1, 1, 1);
        dim3 grid_size_elem((n * k + 255) / 256, 1, 1);
        dim3 block_size_elem(256, 1, 1);

        int option;
        printf("Enter option(1. Row-wise 2. Column-wise 3. Element-wise) :\n");
        scanf("%d", &option);
        switch (option)
        {
        case 1:
                multiply_matrices_row<<<grid_size_row, block_size_row>>>(d_a, d_b, d_c, n, m, k);
                hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);
                printf("Matrix multiplication row-wise: \n");
                for (int i = 0; i < n; i++)
                {
                        for (int j = 0; j < k; j++)
                        {
                                printf("%d ", c[i * k + j]);
                        }
                        printf("\n");
                }
                printf("\n");
                break;
        case 2:
                multiply_matrices_col<<<grid_size_col, block_size_col>>>(d_a, d_b, d_c, n, m, k);
                hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);
                printf("Matrix multiplication column-wise: \n");
                for (int i = 0; i < n; i++)
                {
                        for (int j = 0; j < k; j++)
                        {
                                printf("%d ", c[i * k + j]);
                        }
                        printf("\n");
                }
                printf("\n");

                break;
        case 3:
                multiply_matrices_elem<<<grid_size_elem, block_size_elem>>>(d_a, d_b, d_c, n, m, k);
                hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);
                printf("Matrix multiplication element-wise: \n");
                for (int i = 0; i < n; i++)
                {
                        for (int j = 0; j < k; j++)
                        {
                                printf("%d ", c[i * k + j]);
                        }
                        printf("\n");
                }
                printf("\n");

                break;
        }
        free(a);
        free(b);
        free(c);
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        return 0;
}
